
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2019, iQIYI CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

extern "C" {

__global__ void Crop_uchar(hipTextureObject_t uchar_tex,
                           unsigned char *dst,
                           int dst_width, int dst_height, int dst_pitch,
                           int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (unsigned char) tex2D<unsigned char>(uchar_tex, xi, yi);
}

__global__ void Crop_uchar2(hipTextureObject_t uchar2_tex,
                            uchar2 *dst,
                            int dst_width, int dst_height, int dst_pitch,
                            int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (uchar2) tex2D<uchar2>(uchar2_tex, xi, yi);
}

__global__ void Crop_uchar4(hipTextureObject_t uchar4_tex,
                            uchar4 *dst,
                            int dst_width, int dst_height, int dst_pitch,
                            int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (uchar4) tex2D<uchar4>(uchar4_tex, xi, yi);
}

__global__ void Crop_ushort(hipTextureObject_t ushort_tex,
                            unsigned short *dst,
                            int dst_width, int dst_height, int dst_pitch,
                            int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (unsigned short) tex2D<unsigned short>(ushort_tex, xi, yi);
}

__global__ void Crop_ushort2(hipTextureObject_t ushort2_tex,
                             ushort2 *dst,
                             int dst_width, int dst_height, int dst_pitch,
                             int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (ushort2) tex2D<ushort2>(ushort2_tex, xi, yi);
}

__global__ void Crop_ushort4(hipTextureObject_t ushort4_tex,
                             ushort4 *dst,
                             int dst_width, int dst_height, int dst_pitch,
                             int left, int top)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;
    int xi = xo + left;
    int yi = yo + top;

    if (yo < dst_height && xo < dst_width)
        dst[yo*dst_pitch+xo] = (ushort4) tex2D<ushort4>(ushort4_tex, xi, yi);
}

}